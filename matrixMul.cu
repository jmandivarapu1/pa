#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication as described in Chapter 3
 * of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Supercomputing (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11.
 */

#include <stdio.h>
#include <string>
#include <opencv2/opencv.hpp>

using namespace cv;
using namespace std;

/**
* image: Image to be blurred
* i: Pixel position y
* j: Pixel position x
* return average channel vector
**/

__device__
void get_average(const int &y, const int &x, int *image, int *result, const int &height, const int &width, const int &kernel){
    int blue, green, red, dy, dx, cont, limit;
    blue = green = red = cont = 0;
    
    // Limit of adjacent positions
    limit = kernel / 2;

    // Traverse all adjacents positions
    for( int i = -limit; i <= limit; ++i ){
        for( int j = -limit; j <= limit; ++j ){
            dy = y + i;
            dx = x + j;

            // Verify if point is inside of image
            if( 0 <= dy && dy < height && 0 <= dx && dx < width ){
                blue += image[dy * width * 3 + dx * 3 + 0];
                green += image[dy * width * 3 + dx * 3 + 1];
                red += image[dy * width * 3 + dx * 3 + 2];
                ++cont;
            }
        }
    }

    // Calculate average of each channel
    blue /= cont;
    green /= cont;
    red /= cont;
	
	result[y * width * 3 + x * 3 + 0] = blue;
	result[y * width * 3 + x * 3 + 1] = green;
	result[y * width * 3 + x * 3 + 2] = red;
}

__global__
void blur(int *image, int *result, int *height, int *width, int *kernel) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int n = *height * *width;
    // Calculate pixel average for every pixel   
    int i, j;
    for(int k = index; k < n; k += (stride++)) {
    	i = k / *width;
    	j = k % *width;
    	get_average(i, j, image, result, *height, *width, *kernel);
    }
}

int main(int argc, char** argv){

	Mat image, result,result_pyramid;
	
	int *height_d;
	int *width_d;
	int *kernel_d;
	
	int height;
	int width;
	int kernel;
	int num_threads;
	int num_blocks;

    hipMalloc(&height_d, sizeof(int));
    hipMalloc(&width_d, sizeof(int));
    hipMalloc(&kernel_d, sizeof(int));
  
    if (argc != 5){
        printf("Use: ./blurSecuencial.out <img_path> <num_kernel> <num_threads> <num_blocks>\n");
        return -1;
    }

    image = imread(argv[1], 1);

    if ( !image.data ){
        printf("Image couldn't be read. \n");
        return -1;
    }

    kernel = atoi(argv[2]);
    num_threads = atoi(argv[3]);
    num_blocks = atoi(argv[4]);

    height = image.size().height;
    width = image.size().width;
	
	hipMemcpy(kernel_d, &kernel, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(width_d, &width, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(height_d, &height, sizeof(int), hipMemcpyHostToDevice);

	int *image_mat_d;
	int *image_mat = (int *) malloc (height * width * 3 * sizeof(int));
	
	int *result_mat_d;
	int *result_mat = (int *) malloc (height * width * 3 * sizeof(int));

  	hipMalloc(&image_mat_d, sizeof(int) * height * width * 3);
        hipMalloc(&result_mat_d, sizeof(int) * height * width * 3);
	
    // Create result image with same dimensions
    result = Mat(height, width, CV_8UC3);
    result_pyramid=Mat(height/2,width/2,CV_8UC3);
    
    for (int i = 0; i < height; i++) {
    	for (int j = 0; j < width; j++) {
    		for (int k = 0; k < 3; ++k) {
    			image_mat[i * width * 3 + j * 3 + k] = image.at<Vec3b>(i, j)[k];
    		}
    	}
    }

   
    hipMemcpy(image_mat_d, image_mat, sizeof(int) * height * width * 3, hipMemcpyHostToDevice);
    
//	int num_threads = 256;
//	int num_blocks = (height * width + num_threads - 1) / num_threads;

    blur<<<num_blocks, num_threads>>>(image_mat_d, result_mat_d, height_d, width_d, kernel_d);

    hipMemcpy(result_mat, result_mat_d, sizeof(int) * height * width * 3, hipMemcpyDeviceToHost);

    printf("%d--%d\n",height,width);
    for (int i = 0; i < height; ++i) {
	    for (int j = 0; j < width; ++j) {
		    for (int k = 0; k < 3; ++k) {
			       // printf("%d--%d--%d--%d\n",i,j,k,(i * (width) * 3 + j * 3 + k));
				result.at<Vec3b>(i, j)[k] = result_mat[i * (width) * 3 + j * 3 + k];
		    }
	    }
    }
   printf("%d--%d\n",height,width);

   for (int i = 0; i < height-1;++i) {
            for (int j = 0; j < width-1; ++ j) 
                 {
 // printf("%d--%d--%d--%d--%d\n",i,j,k,(i * (width) * 3 + j * 3 + k),(i * (width) * 3 + (j+1) * 3 + k));
                    printf("%d--%d\n",i,j);
                    for (int k = 0; k < 3; ++k) {
             printf("%d--%d--%d--%d--%d--%d--%d\n",i,j,k,(i/2),(j/2),(i * (width) * 3 + j * 3 + k),(i * (width) * 3 + (j+1) * 3 + k));
                            result_pyramid.at<Vec3b>((i/2), (j/2))[k] = (result_mat[i * (width) * 3 + j * 3 + k]
                                                                        +result_mat[(i) * (width) * 3 + (j+1)* 3 + (k)])/2;
                    }
            }
    }

    printf("%s","its finished the process");
    Mat3b imageF_8UC3;
    result.convertTo(imageF_8UC3, CV_8UC3, 255);
    
    waitKey(0);
    string file_name,file_name_py;
    file_name = "blurred_out.png";
    file_name_py="test_4.jpg";
    //cv ::imshow("test.jpg",result);
    imwrite(file_name, result);
    imwrite(file_name_py, result_pyramid); 
    free(image_mat);
    free(result_mat);
    return 0;
}


//nvcc promising.cu `pkg-config --cflags --libs opencv` -o blur-effect.out
//./blur-effect.out img1.jpg  9 256 652
